// %%writefile dijkstra_cuda_v1.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <limits.h>
#include <assert.h>

//#define DEBUG

static unsigned long int next = 1;

int my_rand(void) {
	return ((next = next * 1103515245 + 12345) % ((u_long) RAND_MAX + 1));
}

void my_srand(unsigned int seed) {
	next = seed;
}

typedef struct {
  int nNodes;   //nro nós
  int *nEdges;  //nro arestas de cada nó.         0 <= i <= nNode
  int **edges;  //arestas
  int **w;      //pesos
} Graph;

void createRandomGraphCUDA(int nNodes, int nEdges, int seed, Graph *graph){
  my_srand(seed); 

  graph->nNodes = nNodes;
  
  hipMallocManaged(&(graph->nEdges), sizeof(int) * nNodes);
  hipMallocManaged(&(graph->edges), sizeof(int*) * nNodes);
  hipMallocManaged(&(graph->w), sizeof(int*) * nNodes);
  
  int k, v;
  for (v = 0; v < nNodes; v++){
    hipMallocManaged(&(graph->edges[v]), sizeof(int) * nNodes);
    hipMallocManaged(&(graph->w[v]), sizeof(int) * nNodes);
    graph->nEdges[v] = 0;
  }

  int source = 0;

  for (source = 0; source < nNodes; source++){
    int nArestasVertice = (double) nEdges / nNodes * (0.5 + my_rand() / (double) RAND_MAX);
    for (k = nArestasVertice; k >= 0; k--){
      int dest = my_rand() % nNodes;
      int w = 1 + (my_rand() % 10);
      graph->edges[source][graph->nEdges[source]] = dest;
      graph->w[source][graph->nEdges[source]++] = w;
    }
  }
}

__global__ void dijkstra_kernel(Graph *d_graph, int min, int *distances){
  int idx = threadIdx.x;
  if (idx < d_graph->nEdges[min]){
    int dest = d_graph->edges[min][idx];
    if (distances[dest] > distances[min] + d_graph->w[min][idx]){
      distances[dest] = distances[min] + d_graph->w[min][idx];
    }
  }
}

void printaMatriz( int** mat, int tam){
  for( int i = 0; i < tam; i++){
    for( int j = 0; j < tam; j++)
      printf("%d ", mat[i][j]);
    printf("\n");
  }
  return;
}

void printaVetor( int* vet, int tam){
  for( int i = 0; i < tam; i++){
    if( i % 80 == 0)
      printf("\n");
    printf("%d ", vet[i]);
  }
  printf("\n");
}


int main(int argc, char **argv){
  int nNodes;
  int nEdges;
  int seed;

  if (argc == 4){
    nNodes = atoi(argv[1]);   //NRO NÓS
    nEdges = atoi(argv[2]);   //MÉDIA DO NRO DE ARESTAS por nó
    seed = atoi(argv[3]);     //SEED PARA GERAÇÃO DO GRAFO
  }else{
    fscanf(stdin, "%d %d %d", &nNodes, &nEdges, &seed);
  }

  nEdges = nNodes * nEdges;
  
  int* visited;
  int* distances;

  hipMallocManaged(&visited, nNodes * sizeof(int));
  hipMallocManaged(&distances, nNodes * sizeof(int));

  Graph *d_graph;
  hipMallocManaged(&d_graph, sizeof(Graph));  
 
  if (visited == NULL || distances == NULL || d_graph == NULL){
    printf("ERRO!!!");
    return 1;
  }

  createRandomGraphCUDA(nNodes, nEdges, seed, d_graph);

  int k = 0, v = 0, source = 0;

  for (v=0; v < nNodes; v++){
    distances[v] = INT_MAX;
    visited[v] = 0;
  }
  
  distances[source] = 0;
  visited[source] = 1;

  for (k=0; k < d_graph->nEdges[source]; k++)
    if(distances[d_graph->edges[source][k]] > d_graph->w[source][k] )
      distances[d_graph->edges[source][k]] = d_graph->w[source][k];


  for (v=1; v < nNodes; v++){
    int min = 0;
    int minValue = INT_MAX;
    for (k=0; k < nNodes; k++)
      if (visited[k] == 0 && distances[k] < minValue){
        minValue = distances[k];
        min = k;
      }
    visited[min] = 1;
    
    dijkstra_kernel<<<1, d_graph->nEdges[min]>>>(d_graph, min, distances);
    hipDeviceSynchronize();
  }

  double mean = 0;
  for (v=0; v < d_graph->nNodes; v++)
    mean += distances[v];
  
  printf("%.2f\n", mean / nNodes);

#ifdef DEBUG
  printf("\nnEdges: ");
  printaVetor(d_graph->nEdges, nNodes);
  printf("\n");

  printf("\nWeights: ");
  printaMatriz(d_graph->w, nNodes);
  printf("\n");

  printf("\nMatriz: \n");
  printaMatriz(d_graph->edges, nNodes);
  printf("\n");

  printf("\nDistancias: ");
  printaVetor(distances, nNodes);
  printf("\n");
#endif

  hipFree(distances);
  hipFree(visited);
  hipFree(d_graph);
  return 0;
}